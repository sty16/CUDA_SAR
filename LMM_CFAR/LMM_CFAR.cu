#include "hip/hip_runtime.h"
#include<mat_opt.h>
#include<stdio.h>
#include<fstream>
#include<stdint.h>
#include<assert.h>
#include<time.h>
#include<unistd.h>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<thrust/sort.h>
#include""
#include<hip/hip_runtime_api.h>
#include<helper_functions.h>

#define MAX_DEPTH       16
#define INSERTION_SORT  32
using namespace cv;


__device__ void  Memcpy(double *im, double *data, int row, int col, int r_c, int r_g, int n);
__device__ void selection_sort(double *data, int left, int right);
__device__ void simple_quicksort(double *data, int left, int right, int depth);


__global__ void lognormal_mixture(double *im, int r_c, int r_g, int k, double Pf, int m, int n) 
{
    /***********************************************************************
    Ship detection based on lognormal mixture models
    INPUT
        im: padding SAR density image
        r_c: radius of the reference window
        r_g: radius of the guard area
        K :number of components
        Pf: false alarm rate
        m : number of rows of input image
        n : number of columns of input image
    OUTPUT
        im_prob:  the cdf of simulate distribution with the im value
    *************************************************************************/
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    int size = r_c*r_c - r_g*r_g;
    for(int i = 0;i<size;i++)
    {
        // data[i] = 1.0;
        // printf("%.1f\n", data[i]);
    }
}
__global__ void malloc_global(double **a)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    if(i==10&&j==10)
    {
        int N = 10000;
        printf("ok");
        hipMalloc((void**)a, sizeof(double)*N);
        for(int i=0;i<N;i++)
        {
            (*a)[i] = i;
        } 
    }
    __syncthreads();
    if(i==11&&j==11)
    {
        printf("%f\n",(*a)[500]);
    }
}
__global__ void CFAR_Gamma(double *im, double *T, int r_c, int r_g, int m, int n) {
    // n_pad为填充后图像的列数， n为原图像的列数
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    int col = threadIdx.y + blockDim.y * blockIdx.y;
    int size = (r_c*r_c-r_g*r_g)*4;int n_pad = n + 2*r_c;
    double clutter_sum = 0, I_C = 0, I = 0, *clutter;
    __shared__  double data[4600];
    if(row < m && col < n)
    {
        int index = threadIdx.x + threadIdx.y*blockDim.x;
        row = row + r_c; col = col + r_c; // 延拓后数据的索引位置发生改变
        clutter =  &data[index*size];
        Memcpy(im, clutter, row, col, r_c, r_g, n_pad);
        simple_quicksort(clutter, 0, size-1, 0);
        int number = size * 0.65;
        for(int i = 0; i< number; i++)
        {
            clutter_sum += clutter[i];
        }
        I_C = clutter_sum/number;
        for(int i = row-1; i <= row+1;i++)
        {
            for(int j = col-1;j <= col+1;j++)
            {
                I += im[i*n_pad+col];
            }
        }
        I = I/9;
        T[(row-r_c)*n+(col-r_c)] = I/I_C; 
        // double *a;
        // hipMalloc((void **)&a,sizeof(double)*1000);
        // for(int i=0;i<1000;i++)
        // {
        //     a[i] = i;
        //     if(i==50)
        //         {printf("ok");printf("%f ", a[i]);}
        // }
        if(row==30&&col==30)
        {
            // for(int i=0;i<size;i++)
            // {
            //     printf("%f ", clutter[i]);
            // }
            // printf("ok");
        }
        // hipFree(a);
    }
}
__device__ void  Memcpy(double *im, double *data, int row, int col, int r_c, int r_g, int n)
{   
    //上部杂波 5x30
    int index = 0;
    for(int i = row-r_c;i<row-r_g;i++)
    {
        for(int j=col-r_c;j<=col+r_c;j++)
        {
            data[index] = im[i*n+j];   
            index += 1;
        }
    }
    //下部杂波 5x30
    for(int i = row+r_g+1;i<=row+r_c;i++)
    {
        for(int j=col-r_c;j<=col+r_c;j++)
        {
            data[index] = im[i*n+j];   
            index += 1;
        }
       
    }
    //左侧杂波20x5
    for(int i = row-r_g;i<=row+r_g;i++)
    {
        for(int j = col-r_c;j<col-r_g;j++)
        {
            data[index] = im[i*n+j];
            index += 1;
        }
    }
    //右侧杂波20x5
    for(int i = row-r_g;i<=row+r_g;i++)
    {
        for(int j = col+r_g+1;j<=col+r_c;j++)
        {
            data[index] = im[i*n+j];
            index += 1;
        }
    }
}

__device__ void selection_sort(double *data, int left, int right)
{
   for(int i = left; i <= right; i++)
   {
        double min_val = data[i]; 
        int min_idx = i;
        for(int j = i+1; j <= right; j++)
        {
            double val_j = data[j];
            if(val_j < min_val)
            {
                min_idx = j;
                min_val = val_j;
            }
        }
        if(i != min_idx)
        {
            data[min_idx] = data[i];
            data[i] = min_val;
        }
   } 
}

__device__  void simple_quicksort(double *data, int left, int right, int depth)
{
    if (depth >= MAX_DEPTH || right-left <= INSERTION_SORT)
    {
        selection_sort(data, left, right);
        return;
    }
    double *lptr = &data[left];
    double *rptr = &data[right];
    double  pivot = data[(left+right)/2];
    while(lptr <= rptr)
    {
        double lval = *lptr;
        double rval = *rptr;
        while(lval < pivot)
        {
            lptr++;
            lval = *lptr;
        }
        while(rval > pivot)
        {
            rptr--;
            rval = *rptr;
        }
        if(lptr <= rptr)
        {
            *lptr++ = rval;
            *rptr-- = lval;
        }
    }
    int nright = rptr - data;
    int nleft  = lptr - data;
    if (left < (rptr-data))
    {
        simple_quicksort(data, left, nright, depth+1);
    }

    // Launch a new block to sort the right part.
    if ((lptr-data) < right)
    {
        simple_quicksort(data, nleft, right, depth+1);
    }
}

int main(int argc, char *argv[])
{
    double **im, *im_pad, *im_dev, *T, *result, threshold;
    int ch, opt_index, channels,m,n;    // opt_index为选项在long_options中的索引
    const char *optstring = "d:c:g:";
    int r_c = 15, r_g = 10;threshold = 4.7;
    dim3D arraydim;
    const char *filename = "../data/data.bin";   
    clock_t start,end;
    start = clock();
    static struct option long_options[] = {
        {"rc", required_argument, NULL,'c'},
        {"rg", required_argument, NULL,'g'}
    };
    while((ch = getopt_long(argc, argv, optstring, long_options, &opt_index)) != -1)
    {
        switch(ch)
        {
            case 'd':
                filename = optarg; break;
            case 'c':
                r_c = atoi(optarg); break;
            case 'g':
                r_g = atoi(optarg); break;
            case '?':
                cout<<"Unknown option: "<<(char)optopt<<endl;
                break;
        }
    }
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    for(int i=0;i<deviceCount;i++)
    {
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp,i);
        cout << "GPU device:" << i << ": " << devProp.name <<endl;
        cout << "global memory: " << devProp.totalGlobalMem / 1024 / 1024 << "MB" <<endl;
        cout << "SM number:" << devProp.multiProcessorCount <<endl;
        cout << "shared memory:" << (devProp.sharedMemPerBlock / 1024.0) <<"KB"<<endl;
        cout << "block max_thread:" << devProp.maxThreadsPerBlock <<endl;
        cout << "registers per Block:" << devProp.regsPerBlock <<endl;
        cout << "SM max theads:" << devProp.maxThreadsPerMultiProcessor <<endl;
        printf("GPU device has compute capabilities (SM %d.%d)\n", devProp.major, devProp.minor);
        cout << "======================================================" <<endl;     
    }
    ifstream infile(filename, ios::in | ios::binary);
    infile.read((char *)&channels, sizeof(int));
    infile.read((char *)&arraydim.m,sizeof(size_t));
    infile.read((char *)&arraydim.n,sizeof(size_t));
    m = (int)arraydim.m; n = (int)arraydim.n;
    im = new double *[m];
    for(int i=0;i<m;i++)
    {
        im[i] = new double[n];
        for(int j=0;j<n;j++)
        {
            infile.read((char *)&im[i][j], sizeof(double));
        }
    }
    Mat image = ArrayToImage(im, arraydim);
    Mat origin_image = ArrayToMat(im, arraydim);
    // double data[3][3] = { {1,2,3},{4,5,6},{7,8,9} };
    Mat pad_image = PadArray(origin_image,r_c,r_c);
    im_pad = pad_image.ptr<double>(0); 
    int row_pad = pad_image.rows;int col_pad = pad_image.cols;
    dim3 blockdim(3,3);
    dim3 griddim((m+blockdim.x-1)/blockdim.x , (n+blockdim.y-1)/blockdim.y);
    checkCudaErrors(hipMalloc((void**)&im_dev, sizeof(double)*row_pad*col_pad));
    checkCudaErrors(hipMalloc((void**)&T, sizeof(double)*m*n));
    checkCudaErrors(hipMemcpy(im_dev, im_pad, sizeof(double)*row_pad*col_pad, hipMemcpyHostToDevice));
    result = new double[m*n];
    hipStream_t detect;
    hipStreamCreate(&detect);
    CFAR_Gamma<<<griddim, blockdim, 0, detect>>>(im_dev, T, r_c, r_g, m, n); //应该传入未填充的图像长宽系数
    // double **a;
    // checkCudaErrors(hipMalloc((void**)&a, sizeof(double *)));
    // griddim.x = 6;griddim.y = 6;
    // malloc_global<<<griddim,blockdim,0>>>(a);
    hipStreamSynchronize(detect);
    checkCudaErrors(hipMemcpy(result, T,  sizeof(double)*m*n, hipMemcpyDeviceToHost));
    Mat detect_result = Mat::zeros(m, n, CV_8UC1);
    for(int i = 0;i<m;i++)
    {
        for(int j = 0;j<n;j++)
        {
            if(result[i*n+j]>threshold)
                detect_result.at<uchar>(i,j) = (unsigned char)255;
            else
                detect_result.at<uchar>(i,j) = (unsigned char)0;
        }
    }
    hipStreamDestroy(detect);
    end = clock();
    imshow("origin" , image); 
    imshow("detected" , detect_result);
    while(char(waitKey())!='q') 
	{    
	}
    // FreeDoubleArray(im,arraydim);
    image.release();
    cout<<"GPU用时："<<(float)(end-start)/CLOCKS_PER_SEC<<end<<endl;
	return 0 ;
}

